
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
 
// CUDA kernel to add elements of two arrays
__global__
void add(int n, float *x, float *y)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;
  for (int i = index; i < n; i += stride)
    y[i] = x[i] + y[i];
}
 
int main(void)
{
  int N = 1<<20;
  float *x, *y;
  float *d_x, *d_y;
 
  int size = N*sizeof(float);

  // allocate memory on both host and device
  x = (float *)malloc(size);
  y = (float *)malloc(size);
  hipMalloc((void **)&d_x, size);
  hipMalloc((void **)&d_y, size);
 
  // initialize x and y arrays on the host
  for (int i = 0; i < N; i++) {
    x[i] = 1.0f;
    y[i] = 2.0f;
  }
 
  // Copy inputs to device
  hipMemcpy(d_x, x, size, hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, size, hipMemcpyHostToDevice);

  // Launch kernel on 1M elements on the GPU
  int blockSize = 256;
  int numBlocks = (N + blockSize - 1) / blockSize;
  add<<<numBlocks, blockSize>>>(N, d_x, d_y);
 
 
  hipMemcpy(y, d_y, size, hipMemcpyDeviceToHost);

  // Check for errors (all values should be 3.0f)
  float maxError = 0.0f;
  for (int i = 0; i < N; i++)
    maxError = fmax(maxError, fabs(y[i]-3.0f));
  std::cout << "Max error: " << maxError << std::endl;
 
  // Free memory
  free(x);
  free(y);
  hipFree(d_x);
  hipFree(d_y);
 
  return 0;
}
